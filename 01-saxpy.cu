#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 200,000 ns.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
    int stride = blockDim.x * gridDim.x;

    for (int i = blockIdx.x * blockDim.x * threadIdx.x; i < N; i += stride)
    {
        c[i] = 2 * a[i] + b[i];
    }
}

int main()
{
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }

    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    int multiProcessorCount = props.multiProcessorCount;
    int warpSize = props.warpSize;

    int threads_per_block = warpSize * 8;
    int number_of_blocks = multiProcessorCount * 32;

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );

    hipDeviceSynchronize();

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
